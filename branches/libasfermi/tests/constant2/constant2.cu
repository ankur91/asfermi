#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char** argv) 
{
	// Create device context.
	hipDevice_t device;
	hipCtx_t context;
	hipError_t cuerr = hipInit(0);
	assert(cuerr == hipSuccess);
	cuerr = hipDeviceGet(&device, 0);
	assert(cuerr == hipSuccess);
	cuerr = hipCtxCreate(&context, hipDeviceScheduleSpin, device);
	assert(cuerr == hipSuccess);

	// Create an output value buffer.
	int* value;
	cuerr = hipMalloc((hipDeviceptr_t*)&value, sizeof(int));
	assert(cuerr == hipSuccess);

	// Load module.
	hipModule_t module;
	cuerr = hipModuleLoad(&module, "constant2.cubin");
	assert(cuerr == hipSuccess);

	// Load the unnamed constant (that is given a name, anyway).
	hipDeviceptr_t unnamed;
	size_t szunnamed = 0;
	cuerr = hipModuleGetGlobal(&unnamed, &szunnamed, module,
		"unnamedConst2_0");
	assert(cuerr == hipSuccess);
	printf("unnamedConst2_0 size = %zu\n", szunnamed);

	// Load the named constant.
	hipDeviceptr_t named;
	size_t sznamed = 0;
	cuerr = hipModuleGetGlobal(&named, &sznamed, module,
		"named");
	assert(cuerr == hipSuccess);
	printf("named size = %zu\n", sznamed);

	// Load kernel.
	hipFunction_t kernel;
	cuerr = hipModuleGetFunction(&kernel, module, "kernel");
	assert(cuerr == hipSuccess);
	
	// Configure kernel launch with output buffer parameter.
	cuerr = cuParamSetSize(kernel, 8);
	assert(cuerr == hipSuccess);
	cuerr = cuParamSetv(kernel, 0, &value, 8);
	assert(cuerr == hipSuccess);
	cuerr = cuFuncSetBlockShape(kernel, 1, 1, 1);
	assert(cuerr == hipSuccess);

	// Launch kernel.
	cuerr = cuLaunch(kernel);
	assert(cuerr == hipSuccess);

	// Wait for kernel completion.
	cuerr = hipCtxSynchronize();
	assert(cuerr == hipSuccess);

	// Get result from device memory.
	int cpu_value;
	cuerr = hipMemcpyDtoH(&cpu_value, (hipDeviceptr_t)value, sizeof(int));
	assert(cuerr == hipSuccess);
	printf("initial result = %d\n", cpu_value);

	// Change the value in contant memory and run
	// kernel again to copy it to the output buffer.
	cpu_value = 10;
	cuerr = hipMemcpyHtoD(unnamed, &cpu_value, sizeof(int));
	assert(cuerr == hipSuccess);
	cpu_value = 0;

	// Configure kernel launch with output buffer parameter.
	cuerr = cuParamSetSize(kernel, 8);
	assert(cuerr == hipSuccess);
	cuerr = cuParamSetv(kernel, 0, &value, 8);
	assert(cuerr == hipSuccess);
	cuerr = cuFuncSetBlockShape(kernel, 1, 1, 1);
	assert(cuerr == hipSuccess);

	// Launch kernel.
	cuerr = cuLaunch(kernel);
	assert(cuerr == hipSuccess);

	// Wait for kernel completion.
	cuerr = hipCtxSynchronize();
	assert(cuerr == hipSuccess);

	// Get result from device memory.
	cuerr = hipMemcpyDtoH(&cpu_value, (hipDeviceptr_t)value, sizeof(int));
	assert(cuerr == hipSuccess);
	printf("changed result = %d\n", cpu_value);

	// Free output buffer.
	cuerr = hipFree((hipDeviceptr_t)value);
	assert(cuerr == hipSuccess);

	// Unload module and destroy context.
        cuerr = hipModuleUnload(module);
	assert(cuerr == hipSuccess);
	cuerr = hipCtxDestroy(context);
	assert(cuerr == hipSuccess);

        return 0;
}

